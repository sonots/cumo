#include "hip/hip_runtime.h"
<% if is_int && !is_object %>
typedef double seq_data_t;
<% else %>
typedef dtype seq_data_t;
<% end %>

<% if is_object %>
typedef size_t seq_count_t;
<% else %>
typedef double seq_count_t;
<% end %>

<% unless is_object %>
__global__ void <%="cumo_#{c_iter}_index_kernel"%>(char *p1, size_t* idx1, seq_data_t beg, seq_data_t step, seq_count_t c, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        dtype x = f_seq(beg,step,c+i);
        *(dtype*)(p1+idx1[i]) = x;
    }
}

__global__ void <%="cumo_#{c_iter}_stride_kernel"%>(char *p1, size_t s1, seq_data_t beg, seq_data_t step, seq_count_t c, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        dtype x = f_seq(beg,step,c+i);
        *(dtype*)(p1+(i*s1)) = x;
    }
}

void <%="cumo_#{c_iter}_index_kernel_launch"%>(char *p1, size_t* idx1, seq_data_t beg, seq_data_t step, seq_count_t c, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_index_kernel"%><<<gridDim, blockDim>>>(p1,idx1,beg,step,c,n);
}

void <%="cumo_#{c_iter}_stride_kernel_launch"%>(char *p1, ssize_t s1, seq_data_t beg, seq_data_t step, seq_count_t c, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_stride_kernel"%><<<gridDim, blockDim>>>(p1,s1,beg,step,c,n);
}
<% end %>
