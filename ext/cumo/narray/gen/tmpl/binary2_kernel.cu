#include "hip/hip_runtime.h"
<% unless type_name == 'robject' %>
__global__ void <%="cumo_#{c_iter}_stride_kernel"%>(char *p1, char *p2, char *p3, char *p4, ssize_t s1, ssize_t s2, ssize_t s3, ssize_t s4, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        m_<%=name%>(*(dtype*)(p1+(i*s1)),*(dtype*)(p2+(i*s2)),*(dtype*)(p3+(i*s3)), *(dtype*)(p4+(i*s4)));
    }
}

void <%="cumo_#{c_iter}_stride_kernel_launch"%>(char *p1, char *p2, char *p3, char *p4, ssize_t s1, ssize_t s2, ssize_t s3, ssize_t s4, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_stride_kernel"%><<<gridDim, blockDim>>>(p1,p2,p3,p4,s1,s2,s3,s4,n);
}
<% end %>
