#include "hip/hip_runtime.h"
<% unless type_name == 'robject' %>
__global__ void <%="cumo_#{c_iter}_kernel"%>(char *p1, char *p2, char *p3, ssize_t s1, ssize_t s2, ssize_t s3, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p3 + (i * s3)) = m_pow(*(dtype*)(p1 + (i * s1)), *(dtype*)(p2 + (i * s2)));
    }
}

__global__ void <%="cumo_#{c_iter}_int32_kernel"%>(char *p1, char *p2, char *p3, ssize_t s1, ssize_t s2, ssize_t s3, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p3 + (i * s3)) = m_pow_int(*(dtype*)(p1 + (i * s1)), *(int32_t*)(p2 + (i * s2)));
    }
}

void <%="cumo_#{c_iter}_kernel_launch"%>(char *p1, char *p2, char *p3, ssize_t s1, ssize_t s2, ssize_t s3, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_kernel"%><<<gridDim, blockDim>>>(p1,p2,p3,s1,s2,s3,n);
}

void <%="cumo_#{c_iter}_int32_kernel_launch"%>(char *p1, char *p2, char *p3, ssize_t s1, ssize_t s2, ssize_t s3, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_int32_kernel"%><<<gridDim, blockDim>>>(p1,p2,p3,s1,s2,s3,n);
}
<% end %>
